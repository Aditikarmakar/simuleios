
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <fstream>

int main(){
    // Initializing variables

    int n = 1024;
    hipfftHandle plan1d;
    double2 *h_a, *d_a, *h_b;

    std::ofstream time_out("time_out.dat"), freq_out("freq_out.dat");

    // Allocations / definition
    h_a = (double2 *)malloc(sizeof(double2)*n);
    h_b = (double2 *)malloc(sizeof(double2)*n);
    for (int i = 0; i < n; ++i){
        h_a[i].x = sin(20*2*M_PI*i/n);
        h_a[i].y = 0;
    }

    hipMalloc(&d_a, sizeof(double2)*n);
    hipMemcpy(d_a, h_a, sizeof(double2)*n, hipMemcpyHostToDevice);
    hipfftPlan1d(&plan1d, n, HIPFFT_Z2Z, 1);

    // FFT
    hipfftExecZ2Z(plan1d, d_a, d_a, HIPFFT_FORWARD);

    // Copying back
    hipMemcpy(h_b, d_a, sizeof(double2)*n, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; ++i){
        time_out << h_a[i].x << '\n';
        freq_out << sqrt(h_b[i].x*h_b[i].x + h_b[i].y*h_b[i].y) << '\n';
    }

    time_out.close();
    freq_out.close();

}
