/*------------int_add---------------------------------------------------------//
*
* Purpose: adding integers with the gpu! I am excited! Woo!
*
*-----------------------------------------------------------------------------*/


#include <hip/hip_runtime.h>
#include<iostream> 

__global__ void add(int *a, int *b, int *c){
    *c = *b + *a;
}

using namespace std;

int main(void){
    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);

    // Allocate space on the gpu
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // input values 
    a = 2;
    b = 7;

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    add<<<1,1>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c); 

    cout << "integer from GPU is: " <<  c << endl;

    return 0;
}
